#include "hip/hip_runtime.h"
#ifndef MSD_KERNEL_H
#define MSD_KERNEL_H

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__constant__ float factor0;
__constant__ float factor1;
__constant__ unsigned int N;

texture<float4, hipTextureType2D, hipReadModeElementType> forceTexture;

// function to do time corrected Verlet integration for first two time-steps
extern "C" __global__ void displace_01 (float4 *src, float4 *dest, float2 *texCoords, float *mass)
{
  int tid = blockIdx.x*blockDim.x + threadIdx.x;

  while (tid < N){

    float2 coords = texCoords [tid];
  	float4 force = tex2D (forceTexture, coords.x, coords.y);
  	float lmass_rec = mass [tid];

  	float4 acceleration = {lmass_rec*force.x, lmass_rec*force.y, lmass_rec*force.z, 1.};
  	float4 velocity = {factor0*acceleration.x, factor0*acceleration.y, factor0*acceleration.z, 1.};

    float4 prev = src [tid];
    float4 curr = {
                  prev.x + factor0*velocity.x + 0.5*factor1*acceleration.x,
                  prev.y + factor0*velocity.y + 0.5*factor1*acceleration.y,
                  prev.z + factor0*velocity.z + 0.5*factor1*acceleration.z,
                  1.
                  };
    dest [tid] = curr;

    tid += blockDim.x*gridDim.x;
  }
}

// function to do time corrected Verlet integration for time-steps 3 and later
extern "C" __global__ void displace_n (float4 *src, float4 *dest, float2 *texCoords, float *mass)
{
  int tid = blockIdx.x*blockDim.x + threadIdx.x;

  while (tid < N){

    float2 coords = texCoords [tid];
  	float4 force = tex2D (forceTexture, coords.x, coords.y);
  	float lmass_rec = mass [tid];

  	float4 acceleration = {lmass_rec*force.x, lmass_rec*force.y, lmass_rec*force.z, 1.};

    float4 prev = dest [tid];
    float4 curr = src [tid];

    float4 future = {
                    curr.x + factor0*(curr.x - prev.x) + factor1*acceleration.x,
                    curr.y + factor0*(curr.y - prev.y) + factor1*acceleration.y,
                    curr.z + factor0*(curr.z - prev.z) + factor1*acceleration.z,
                    1.
                    };

    dest [tid] = future;

    tid += blockDim.x*gridDim.x;
  }
}

#endif
