#ifndef XFEM_KERNEL_H
#define XFEM_KERNEL_H

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

extern "C" __global__ void conjugate_gradient (float *src, float *dest, unsigned int N)
{
  unsigned int ind = blockIdx.x*blockDim.x + threadIdx.x;
  if (ind < N){
    dest [ind] = src[ind] + 1.;
  }
}

#endif
